

#include <hip/hip_runtime.h>
#include <stdio.h>


void __global__ mean_pool(float* means, float *words, int *lengths,int *prevLengths, int numdocs, int dims) 
{

    	int bid = blockIdx.x;

	__shared__ float local_means[256];

    	for(int step = bid; step < numdocs; step += gridDim.x )
	{
		int wordsInDoc = lengths[step];
		int blockStarts = prevLengths[step]*dims; 

		local_means[threadIdx.x] = 0.0;

		for (int i = blockStarts + threadIdx.x; i < blockStarts+(wordsInDoc*dims) ; i += dims)
			local_means[threadIdx.x] +=  words[i];
		
		__syncthreads();

		means[step*dims + threadIdx.x] = local_means[threadIdx.x]/(float)wordsInDoc;
	}
}


void __global__ backprop_mean_pool(float* means, float *words, int *lengths,int *prevLengths, int numdocs, int dims) 
{

    	int bid = blockIdx.x;

	__shared__ float local_means[256];


    	for(int step = bid; step < numdocs; step += gridDim.x )
	{
		int wordsInDoc = lengths[step];
		int blockStarts = prevLengths[step]*dims; 

		local_means[threadIdx.x] = means[step*dims+threadIdx.x];

		for (int i = blockStarts + threadIdx.x; i < blockStarts+(wordsInDoc*dims) ; i += dims)
			words[i] = local_means[threadIdx.x]/wordsInDoc;
		
	}
}



void __global__ max_pool(float* maxes, int* which, float *words, int *lengths,int *prevLengths, int numdocs, int dims) 
{
    	int bid = blockIdx.x;

	__shared__ float local_maxes[256];
	__shared__ short local_which[256];

    	for(int step = bid; step < numdocs; step += gridDim.x )
	{
		int wordsInDoc = lengths[step];
		int blockStarts = prevLengths[step]*dims; 

		local_maxes[threadIdx.x] = words[blockStarts+threadIdx.x];
		local_which[threadIdx.x] = 0;
		short j=1;	// the word index in a doc

		for (int i = blockStarts+dims+threadIdx.x; i < blockStarts+(wordsInDoc*dims) ; i += dims)
		{
			if(words[i]>local_maxes[threadIdx.x])
			{
				local_maxes[threadIdx.x] =  words[i];
				local_which[threadIdx.x] = j;
			}
			j++; 
		}
		__syncthreads();

		maxes[step*dims + threadIdx.x] = local_maxes[threadIdx.x];
		which[step*dims + threadIdx.x] = local_which[threadIdx.x];
	}
}

void __global__ backprop_max_pool(float* maxes, int* which, float *words, int *lengths,int *prevLengths, int numdocs, int dims) 
{
    	int bid = blockIdx.x;

	__shared__ float local_maxes[256];
	__shared__ short local_which[256];

    	for(int step = bid; step < numdocs; step += gridDim.x )
	{
		int wordsInDoc = lengths[step];
		int blockStarts = prevLengths[step]*dims; 

		local_maxes[threadIdx.x] = maxes[step*dims+threadIdx.x];
		local_which[threadIdx.x] = which[step*dims+threadIdx.x];
		short j=0;	// the word index in a doc

		for (int i = blockStarts+threadIdx.x; i < blockStarts+(wordsInDoc*dims) ; i += dims)
		{
			if(local_which[threadIdx.x]==j)
			{
				words[i] =  local_maxes[threadIdx.x];
			}
			else
				words[i]=0;
			j++; 
		}

	}
}

