/*
This is the central piece of code, the interface of GPU_ops
This class will get translated into python
*/

#include "kernel.cu"
#include <manager.hh>
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;


GPU_Ops::GPU_Ops (float* means_host_,float* words_host_, float* bProp_words_, float* maxes_host_, int* which_host_, float* backMax_words_host_,
 				int* lengths_host_, int* prevLengths_host_,int numdocs_,int dims_) 
{

  means_host = means_host_;
  words_host = words_host_;
  lengths_host = lengths_host_;
  prevLengths_host = prevLengths_host_; 
  numdocs = numdocs_;
  dims = dims_;
  backPMean_words_host = bProp_words_;
  maxes_host = maxes_host_;
  which_host = which_host_;
  backMax_words_host = backMax_words_host_;
 
  int full_size = (prevLengths_host[numdocs-1]+lengths_host[numdocs-1]) * dims *sizeof(float);
  hipError_t err; 

  err = hipMalloc((void**) &words_device, full_size);
  assert(err == 0);

  err = hipMalloc((void**) &backPMean_words_device, full_size);
  assert(err == 0);
  
  err = hipMalloc((void**) &means_device, numdocs*dims*sizeof(float));
  assert(err == 0);

  err = hipMalloc((void**) &lengths_device, numdocs*sizeof(int));
  assert(err == 0);
  
  err = hipMalloc((void**) &prevLengths_device, numdocs*sizeof(int));
  assert(err == 0);

  err = hipMalloc((void**) &which_device, numdocs*dims*sizeof(int));
  assert(err == 0);
  
  err = hipMalloc((void**) &maxes_device, numdocs*dims*sizeof(float));
  assert(err == 0);

  err = hipMalloc((void**) &backPMax_words_device, full_size);
  assert(err == 0);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  // copying data from HostToDevice
  err = hipMemcpy(words_device, words_host, full_size, hipMemcpyHostToDevice);
  assert(err == 0);

  err = hipMemcpy(lengths_device, lengths_host, numdocs*sizeof(int), hipMemcpyHostToDevice);
  assert(err == 0);

  err = hipMemcpy(prevLengths_device, prevLengths_host, numdocs*sizeof(int), hipMemcpyHostToDevice);
  assert(err == 0);

  mean_pool<<<32, dims>>>(means_device, words_device, lengths_device, prevLengths_device, numdocs, dims);

  //hipDeviceSynchronize();

  err = hipGetLastError();

  if(err != 0) 
  {  cout << "cuda kernel returned error: "<< hipGetErrorString(err) << endl; assert(0); }


  backprop_mean_pool<<<32, dims>>>(means_device, backPMean_words_device, lengths_device, prevLengths_device, numdocs, dims);

  err = hipGetLastError();

  if(err != 0) 
  {  cout << "cuda kernel returned error: "<< hipGetErrorString(err) << endl; assert(0); }

  max_pool<<<32, dims>>>(maxes_device, which_device, words_device, lengths_device, prevLengths_device, numdocs, dims);

  err = hipGetLastError();

  if(err != 0) 
  {  cout << "cuda kernel returned error: "<< hipGetErrorString(err) << endl; assert(0); }

  backprop_max_pool<<<32, dims>>>(maxes_device, which_device, backPMax_words_device, lengths_device, prevLengths_device, numdocs, dims);

  err = hipGetLastError();

  if(err != 0) 
  {  cout << "cuda kernel returned error: "<< hipGetErrorString(err) << endl; assert(0); }

  hipMemcpy(means_host, means_device, numdocs*dims*sizeof(float), hipMemcpyDeviceToHost);
  err = hipGetLastError();
  if(err != 0) 
  	{  cout << hipGetErrorString(err) << endl; assert(0); }

  hipMemcpy(backPMean_words_host, backPMean_words_device, full_size, hipMemcpyDeviceToHost);
  err = hipGetLastError();
  if(err != 0) 
  	{  cout << hipGetErrorString(err) << endl; assert(0); }

  hipMemcpy(which_host, which_device, numdocs*dims*sizeof(int), hipMemcpyDeviceToHost);
  err = hipGetLastError();
  if(err != 0) 
  	{  cout << hipGetErrorString(err) << endl; assert(0); }

  hipMemcpy(backMax_words_host, backPMax_words_device, full_size, hipMemcpyDeviceToHost);
  err = hipGetLastError();
  if(err != 0) 
  	{  cout << hipGetErrorString(err) << endl; assert(0); }

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("time elapsed for gpu_mean_pool(): %f milliseconds\n", milliseconds);

}



GPU_Ops::~GPU_Ops() {
  hipFree(words_device);
  hipFree(backPMean_words_device);
  hipFree(lengths_device);
  hipFree(prevLengths_device);
  hipFree(backPMax_words_device);
  hipFree(means_device);
}
