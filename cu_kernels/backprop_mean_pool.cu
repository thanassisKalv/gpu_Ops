
#include <hip/hip_runtime.h>

/*
	// Cython function from 'thinc' library
class NumpyOps(Ops):

    def backprop_mean_pool(self, float[:, ::1] d_means, int[::1] lengths):
        cdef int B = lengths.shape[0]
        cdef int O = d_means.shape[1]
        cdef int T = 0
        for length in lengths[:B]:
            T += length
        cdef Pool mem = Pool()
        dX = <float*>mem.alloc(T * O, sizeof(float))

        cpu_backprop_mean_pool(dX,
            &d_means[0,0], &lengths[0], B, T, O)

        return cpu_floats_ptr2array(dX, (T, O))

cdef void cpu_backprop_mean_pool(float* dX__to,
        const float* d_means__bo, const int* lengths__b,
        int B, int T, int O) nogil:
    cdef float scale = 0.
    for length in lengths__b[:B]:
        scale = 1./ length
        for _ in range(length):
            VecVec.add_i(dX__to,
                d_means__bo, scale, O)
            dX__to += O
        d_means__bo += O

*/


void __global__ backprop_mean_pool(float* means, float *words, int *lengths,int *prevLengths, int numdocs, int dims) 
{

    	int bid = blockIdx.x;

	__shared__ float local_means[256];


    	for(int step = bid; step < numdocs; step += gridDim.x )
	{
		int wordsInDoc = lengths[step];
		int blockStarts = prevLengths[step]*dims; 

		local_means[threadIdx.x] = means[step*dims+threadIdx.x];

		for (int i = blockStarts + threadIdx.x; i < blockStarts+(wordsInDoc*dims) ; i += dims)
			words[i] = local_means[threadIdx.x]/wordsInDoc;
		
	}
}

