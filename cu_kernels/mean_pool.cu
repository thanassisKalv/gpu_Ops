
#include <hip/hip_runtime.h>


/*
	// Cython function from 'thinc' library
class NumpyOps(Ops):

    def mean_pool(self, float[:, ::1] X, int[::1] lengths):
        cdef int B = lengths.shape[0]
        cdef int O = X.shape[1]
        cdef int T = X.shape[0]

        cdef Pool mem = Pool()
        means = <float*>mem.alloc(B * O, sizeof(float))

        cpu_mean_pool(means,
            &X[0, 0], &lengths[0], B, T, O)
        return cpu_floats_ptr2array(means, (B, O))


cdef void cpu_mean_pool(float* means__bo,
        const float* X__to, const int* lengths__b,
        int B, int T, int O) nogil:
    '''Compute means of a batch of concatenated sequences, using the lengths.'''
    cdef float scale = 0.
    for length in lengths__b[:B]:
        scale = 1. / length
        for _ in range(length):
            VecVec.add_i(means__bo,
                X__to, scale, O)
            X__to += O
        means__bo += O
*/

// hardcoded the shared memory to 256  but we can easily change the host to invoke
// the kernel to dynamically allocate the shared memory (according to vector dimensions) 
void __global__ mean_pool(float* means, float *words, int *lengths,int *prevLengths, int numdocs, int dims) 
{

    	int bid = blockIdx.x;

	__shared__ float local_means[256];

    	for(int step = bid; step < numdocs; step += gridDim.x )
	{
		int wordsInDoc = lengths[step];
		int blockStarts = prevLengths[step]*dims; 

		local_means[threadIdx.x] = 0.0;

		for (int i = blockStarts + threadIdx.x; i < blockStarts+(wordsInDoc*dims) ; i += dims)
			local_means[threadIdx.x] +=  words[i];
		
		__syncthreads();

		means[step*dims + threadIdx.x] = local_means[threadIdx.x]/(float)wordsInDoc;
	}
}


