
#include <hip/hip_runtime.h>

/*
	// Cython function from 'thinc' library
class NumpyOps(Ops):

    def max_pool(self, float[:, ::1] X, int[::1] lengths):
        cdef int B = lengths.shape[0]
        cdef int O = X.shape[1]
        cdef int T = X.shape[0]

        cdef Pool mem = Pool()
        maxes = <float*>mem.alloc(B * O, sizeof(float))
        which = <int*>mem.alloc(B * O, sizeof(int))

        cpu_max_pool(maxes, which,
            &X[0, 0], &lengths[0], B, T, O)

        cdef ndarray py_best = cpu_floats_ptr2array(maxes, (B, O))
        cdef ndarray py_which = cpu_ints_ptr2array(which, (B, O))
        return py_best, py_which

cdef void cpu_max_pool(float* maxes__bo, int* which__bo,
        const float* X__to, const int* lengths__b,
        int B, int T, int O) nogil:
    '''Compute maxes of a batch of concatenated sequences, using the lengths.'''
    cdef float scale = 0.
    for length in lengths__b[:B]:
        memcpy(maxes__bo, X__to, O * sizeof(maxes__bo[0]))
        memset(which__bo, 0, O * sizeof(which__bo[0]))
        X__to += O
        for i in range(1, length):
            for j in range(O):
                if X__to[j] > maxes__bo[j]:
                    maxes__bo[j] = X__to[j]
                    which__bo[j] = i
            X__to += O
        maxes__bo += O
        which__bo += O

*/


void __global__ max_pool(float* maxes, int* which, float *words, int *lengths,int *prevLengths, int numdocs, int dims) 
{
    	int bid = blockIdx.x;

	__shared__ float local_maxes[256];
	__shared__ short local_which[256];

    	for(int step = bid; step < numdocs; step += gridDim.x )
	{
		int wordsInDoc = lengths[step];
		int blockStarts = prevLengths[step]*dims; 

		local_maxes[threadIdx.x] = words[blockStarts+threadIdx.x];
		local_which[threadIdx.x] = 0;
		short j=1;	// the word index in a doc

		for (int i = blockStarts+dims+threadIdx.x; i < blockStarts+(wordsInDoc*dims) ; i += dims)
		{
			if(words[i]>local_maxes[threadIdx.x])
			{
				local_maxes[threadIdx.x] =  words[i];
				local_which[threadIdx.x] = j;
			}
			j++; 
		}
		__syncthreads();

		maxes[step*dims + threadIdx.x] = local_maxes[threadIdx.x];
		which[step*dims + threadIdx.x] = local_which[threadIdx.x];
	}
}
