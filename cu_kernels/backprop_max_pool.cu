
#include <hip/hip_runtime.h>

/*
	// Cython function from 'thinc' library
class NumpyOps(Ops):

    def backprop_max_pool(self, float[:, ::1] d_maxes,
            int[:, ::1] which, int[::1] lengths):
        cdef int B = lengths.shape[0]
        cdef int O = d_maxes.shape[1]
        cdef int T = 0
        for length in lengths[:B]:
            T += length
        cdef Pool mem = Pool()
        dX = <float*>mem.alloc(T * O, sizeof(float))

        cpu_backprop_max_pool(dX,
            &d_maxes[0,0], &which[0, 0], &lengths[0], B, T, O)

        return cpu_floats_ptr2array(dX, (T, O))


cdef void cpu_backprop_max_pool(float* dX__to,
        const float* d_maxes__bo, const int* which__bo, const int* lengths__b,
        int B, int T, int O) nogil:
    cdef int length, i, j
    for length in lengths__b[:B]:
        for i in range(length):
            for j in range(O):
                if which__bo[j] == i:
                    dX__to[j] += d_maxes__bo[j]
            dX__to += O
        d_maxes__bo += O
        which__bo += O
*/


void __global__ backprop_max_pool(float* maxes, int* which, float *words, int *lengths,int *prevLengths, int numdocs, int dims) 
{
    	int bid = blockIdx.x;

	__shared__ float local_maxes[256];
	__shared__ short local_which[256];

    	for(int step = bid; step < numdocs; step += gridDim.x )
	{
		int wordsInDoc = lengths[step];
		int blockStarts = prevLengths[step]*dims; 

		local_maxes[threadIdx.x] = maxes[step*dims+threadIdx.x];
		local_which[threadIdx.x] = which[step*dims+threadIdx.x];
		short j=0;	// the word index in a doc

		for (int i = blockStarts+threadIdx.x; i < blockStarts+(wordsInDoc*dims) ; i += dims)
		{
			if(local_which[threadIdx.x]==j)
			{
				words[i] =  local_maxes[threadIdx.x];
			}
			else
				words[i]=0;
			j++; 
		}

	}
}

